#include <torch/extension.h>
#include <hip/hip_runtime.h>

#include "build_hash_kernel.h"

int build_hash_cuda(const at::Tensor& th_points,
        at::Tensor th_hash_entries,
        at::Tensor th_hash_keys,
        at::Tensor th_neib_ents,
        at::Tensor th_barycentric,
        at::Tensor th_valid_entries,
        at::Tensor th_n_valid,
        size_t hash_cap) {

    const float* points = th_points.data<float>();
    int* hash_entries = th_hash_entries.data<int>();
    short* hash_keys = th_hash_keys.data<short>();
    int* neib_ents = th_neib_ents.data<int>();
    float* barycentric = th_barycentric.data<float>();
    int* valid_entries = th_valid_entries.data<int>();
    int* n_valid = th_n_valid.data<int>();

    const size_t dim = th_points.sizes()[0];
    const size_t N = th_points.sizes()[1] * th_points.sizes()[2];

    hipError_t err;
    call_build_hash_kernels(points, hash_entries, hash_keys, neib_ents, barycentric, valid_entries, n_valid, hash_cap, N, dim, 0);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "build_hash CUDA kernel failure: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    return 1;
}
